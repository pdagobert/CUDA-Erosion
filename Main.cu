#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <chrono>

#include <glm/glm.hpp>
#include <Simplex.h>

std::vector< float > generateNoiseTexture( int size )
{
    std::vector< float > heightmap;
    heightmap.reserve( size * size );

    for( int y = 0; y < size; y++ )
    {
        for( int x = 0; x < size; x++ )
        {
            glm::vec2 position( x, y );
            position /= size;

            float height = Simplex::fBm( position, 8, 2.0f, 0.5f );
            heightmap.push_back( height );
        }
    }

    return heightmap;
}

// https://github.com/RolandR/glterrain/blob/master/js/terrain.js
void erode( std::vector< float >& heightmap, int size, int iterations )
{
    const float Scale = 1.0f;
    const float Erosion = 0.0005 * Scale;
    const float Deposition = 0.0000002 * Scale;
    const float Evaporation = 0.9f;

    std::vector< float > water( heightmap.size(), 1.0f );
    std::vector< float > dWater( heightmap.size(), 0 );
    std::vector< float > dheightV( heightmap.size(), 0 );

    for( int i = 0; i < iterations; i++ )
    {
        // ignore bounds for now
        for( int y = 1; y < size - 2; y++ )
        {
            for( int x = 1; x < size - 2; x++ )
            {
                int index = x + y * size;

                float m = heightmap[ index ];
                float dtl = heightmap[ x - 1 + ( y + 1 ) * size ];
                dtl = glm::max( m - dtl, 0.0f );
                float dt = heightmap[ x + ( y + 1 ) * size ];
                dt = glm::max( m - dt, 0.0f );
                float dtr = heightmap[ x + 1 + ( y + 1 ) * size ];
                dtr = glm::max( m - dtr, 0.0f );
                float dml = heightmap[ x - 1 + y * size ];
                dml = glm::max( m - dml, 0.0f );
                float dmr = heightmap[ x + 1 + y * size ];
                dmr = glm::max( m - dmr, 0.0f );
                float dbl = heightmap[ x - 1 + ( y - 1 ) * size ];
                dbl = glm::max( m - dbl, 0.0f );
                float db = heightmap[ x + ( y - 1 ) * size ];
                db = glm::max( m - db, 0.0f );
                float dbr = heightmap[ x + 1 + ( y - 1 ) * size ];
                dbr = glm::max( m - dbr, 0.0f );

                float dheight = dtl + dt + dtr + dml + dmr + dbl + db + dbr;

                if( dheight != 0.0f )
                {
                    float w = water[ index ] * Evaporation;
                    float remainingWater = w * 0.0002 / ( dheight * Scale + 1.0f );
                    w -= remainingWater;

                    dWater[ x - 1 + ( y + 1 ) * size ] += dtl / dheight * w;
                    dWater[ x + ( y + 1 ) * size ] += dt / dheight * w;
                    dWater[ x + 1 + ( y + 1 ) * size ] += dtr / dheight * w;
                    dWater[ x - 1 + y * size ] += dml / dheight * w;
                    dWater[ x + 1 + y * size ] += dmr / dheight * w;
                    dWater[ x - 1 + ( y - 1 ) * size ] += dbl / dheight * w;
                    dWater[ x + ( y - 1 ) * size ] += db / dheight * w;
                    dWater[ x + 1 + ( y - 1 ) * size ] += dbr / dheight * w;

                    water[ index ] = 1.0f + remainingWater;
                }

                dheightV[ index ] = dheight;
            }
        }

        for( int y = 1; y < size - 2; y++ )
        {
            for( int x = 1; x < size - 2; x++ )
            {
                int index = x + y * size;

                water[ index ] += dWater[ index ];
                dWater[ index ] = 0.0f;

                float oldHeight = heightmap[ index ];
                heightmap[ index ] += ( -( dheightV[ index ] - 0.005f / Scale ) * water[ index ] ) * Erosion + water[ index ] * Deposition;

                if( oldHeight < heightmap[ index ] )
                    water[ index ] = glm::max( water[index ] - ( heightmap[ index ] - oldHeight ) * 1000.0f, 0.0f );
            }
        }
    }
}

void save( const std::string& fileName, int width, int height, const std::vector< float >& heightmap )
{
    std::ofstream file( fileName, std::ios::binary );
    if( file.fail() )
    {
        std::cout << "cannot save file " << fileName << std::endl;
        return;
    }

    file.write( reinterpret_cast< char* >( &width ), sizeof( int ) );
    file.write( reinterpret_cast< char* >( &height ), sizeof( int ) );
    file.write( reinterpret_cast< const char* >( &heightmap[ 0 ] ), heightmap.size() * sizeof( float ) );
}

void cpuErosion( int size, int iterations )
{
    auto heightmap = generateNoiseTexture( size );

    auto start = std::chrono::high_resolution_clock::now();
    erode( heightmap, size, iterations );

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast< std::chrono::milliseconds >( end - start ).count();

    std::cout << "cpu erosion took " << duration << "ms" << std::endl;

    save( "noiseCPU.raw", size, size, heightmap );
}

__global__
void erodeKernel( float* heightmap, int size, int iterations, float* water, float* tmpWater, float* dheightV )
{
    const float Scale = 1.0f;
    const float Erosion = 0.0005 * Scale;
    const float Deposition = 0.0000002 * Scale;
    const float Evaporation = 0.9f;

    int startx = threadIdx.x + blockIdx.x * blockDim.x;
    int starty = threadIdx.y + blockIdx.y * blockDim.y;
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;

    for( int i = 0; i < iterations; i++ )
    {
        for( int y = 1 + starty; y < size - 1; y += stridey )
        {
            for( int x = 1 + startx; x < size - 1; x += stridex )
            {
                int index = x + y * size;

                float m = heightmap[ index ];

                float dtl = heightmap[ x - 1 + ( y + 1 ) * size ];
                dtl = glm::max( m - dtl, 0.0f );
                float dt = heightmap[ x + ( y + 1 ) * size ];
                dt = glm::max( m - dt, 0.0f );
                float dtr = heightmap[ x + 1 + ( y + 1 ) * size ];
                dtr = glm::max( m - dtr, 0.0f );
                float dml = heightmap[ x - 1 + y * size ];
                dml = glm::max( m - dml, 0.0f );
                float dmr = heightmap[ x + 1 + y * size ];
                dmr = glm::max( m - dmr, 0.0f );
                float dbl = heightmap[ x - 1 + ( y - 1 ) * size ];
                dbl = glm::max( m - dbl, 0.0f );
                float db = heightmap[ x + ( y - 1 ) * size ];
                db = glm::max( m - db, 0.0f );
                float dbr = heightmap[ x + 1 + ( y - 1 ) * size ];
                dbr = glm::max( m - dbr, 0.0f );

                float dheight = dtl + dt + dtr + dml + dmr + dbl + db + dbr;

                if( dheight != 0.0f )
                {
                    float w = water[ index ] * Evaporation;
                    float remainingWater = w * 0.0002 / ( dheight * Scale + 1.0f );
                    w -= remainingWater;

                    // the only place where race condition can occur

                    atomicAdd( &tmpWater[ x - 1 + ( y + 1 ) * size ], dtl / dheight * w );
                    atomicAdd( &tmpWater[ x + ( y + 1 ) * size ], dt / dheight * w );
                    atomicAdd( &tmpWater[ x + 1 + ( y + 1 ) * size ], dtr / dheight * w );
                    atomicAdd( &tmpWater[ x - 1 + y * size ], dml / dheight * w );
                    atomicAdd( &tmpWater[ x + 1 + y * size ], dmr / dheight * w );
                    atomicAdd( &tmpWater[ x - 1 + ( y - 1 ) * size ], dbl / dheight * w );
                    atomicAdd( &tmpWater[ x + ( y - 1 ) * size ], db / dheight * w );
                    atomicAdd( &tmpWater[ x + 1 + ( y - 1 ) * size ], dbr / dheight * w );

                    water[ index ] = 1.0f + remainingWater;
                }

                dheightV[ index ] = dheight;
            }
        }

        __syncthreads();

        for( int y = 1 + starty; y < size - 1; y += stridey )
        {
            for( int x = 1 + startx; x < size - 1; x += stridex )
            {
                int index = x + y * size;

                water[ index ] += tmpWater[ index ];
                tmpWater[ index ] = 0;

                float oldHeight = heightmap[ index ];
                heightmap[ index ] += ( -( dheightV[ index ] - 0.005f / Scale ) * water[ index ] ) * Erosion + water[ index ] * Deposition;

                if( oldHeight < heightmap[ index ] )
                    water[ index ] = glm::max( water[index ] - ( heightmap[ index ] - oldHeight ) * 1000.0f, 0.0f );
            }
        }

        __syncthreads();
    }
}

__global__
void erodeFirstPassKernel( float* __restrict__ heightmap, int size, float* __restrict__ water, float* __restrict__ tmpWater, float* __restrict__ dheightV )
{
    __shared__ float heightmapCache[ 34 * 34 ];

    const float Scale = 1.0f;
    const float Evaporation = 0.9f;

    /*int startx = threadIdx.x + blockIdx.x * blockDim.x;
    int starty = threadIdx.y + blockIdx.y * blockDim.y;
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;

    for( int y = 1 + starty; y < size - 2; y += stridey )
    {
        for( int x = 1 + startx; x < size - 2; x += stridex )
        {*/
            int x2 = threadIdx.x + blockIdx.x * blockDim.x;
            int y2 = threadIdx.y + blockIdx.y * blockDim.y;
            int indexGlobal = x2 + y2 * size;

            int tileSize = 34;

            int x = threadIdx.x + 1;
            int y = threadIdx.y + 1;
            int indexLocal = x + y * tileSize;

            heightmapCache[ indexLocal ] = heightmap[ indexGlobal ];

            if( x2 == 0 || y2 == 0 || x2 == size - 1 || y2 == size - 1 )
                return;

            int offset = 0;

            if( threadIdx.x == 0 )
                offset = -1;

            if( threadIdx.x == 31 )
                offset = 1;

            if( offset != 0 )
            {
                int borderGlobal = x2 + offset + y2 * size;
                int borderLocal = x + offset + y * tileSize;
                heightmapCache[ borderLocal ] = heightmap[ borderGlobal ];
            }

            offset = 0;

            if( threadIdx.y == 0 )
                offset = -1;

            if( threadIdx.y == 31 )
                offset = 1;

            if( offset != 0 )
            {
                int borderGlobal = x2 + ( y2 + offset ) * size;
                int borderLocal = x + ( y + offset ) * tileSize;
                heightmapCache[ borderLocal ] = heightmap[ borderGlobal ];
            }

            int xoffset = 0;
            int yoffset = 0;

            if( threadIdx.x == 0 && threadIdx.y == 0 )
            {
                xoffset = -1;
                yoffset = -1;
            }

            if( threadIdx.x == 31 && threadIdx.y == 0 )
            {
                xoffset = 1;
                yoffset = -1;
            }

            if( threadIdx.x == 0 && threadIdx.y == 31 )
            {
                xoffset = -1;
                yoffset = 1;
            }

            if( threadIdx.x == 31 && threadIdx.y == 31 )
            {
                xoffset = 1;
                yoffset = 1;
            }

            if( xoffset != 0 )
            {
                int borderGlobal = x2 + xoffset + ( y2 + yoffset ) * size;
                int borderLocal = x + xoffset + ( y + yoffset ) * tileSize;
                heightmapCache[ borderLocal ] = heightmap[ borderGlobal ];
            }

            __syncthreads();

            float m = heightmapCache[ x + y * tileSize ];
            float dtl = heightmapCache[ x - 1 + ( y + 1 ) * tileSize ];
            dtl = max( m - dtl, 0.0f );
            float dt = heightmapCache[ x + ( y + 1 ) * tileSize ];
            dt = max( m - dt, 0.0f );
            float dtr = heightmapCache[ x + 1 + ( y + 1 ) * tileSize ];
            dtr = max( m - dtr, 0.0f );
            float dml = heightmapCache[ x - 1 + y * tileSize ];
            dml = max( m - dml, 0.0f );
            float dmr = heightmapCache[ x + 1 + y * tileSize ];
            dmr = max( m - dmr, 0.0f );
            float dbl = heightmapCache[ x - 1 + ( y - 1 ) * tileSize ];
            dbl = max( m - dbl, 0.0f );
            float db = heightmapCache[ x + ( y - 1 ) * tileSize ];
            db = max( m - db, 0.0f );
            float dbr = heightmapCache[ x + 1 + ( y - 1 ) * tileSize ];
            dbr = max( m - dbr, 0.0f );

            float dheight = dtl + dt + dtr + dml + dmr + dbl + db + dbr;

            if( dheight != 0.0f )
            {
                float w = water[ indexGlobal ] * Evaporation;
                float remainingWater = w * 0.0002 / ( dheight * Scale + 1.0f );
                w -= remainingWater;

                // the only place where race condition can occur

                atomicAdd( &tmpWater[ x2 - 1 + ( y2 + 1 ) * size ], dtl / dheight * w );
                atomicAdd( &tmpWater[ x2 + ( y2 + 1 ) * size ], dt / dheight * w );
                atomicAdd( &tmpWater[ x2 + 1 + ( y2 + 1 ) * size ], dtr / dheight * w );
                atomicAdd( &tmpWater[ x2 - 1 + y2 * size ], dml / dheight * w );
                atomicAdd( &tmpWater[ x2 + 1 + y2 * size ], dmr / dheight * w );
                atomicAdd( &tmpWater[ x2 - 1 + ( y2 - 1 ) * size ], dbl / dheight * w );
                atomicAdd( &tmpWater[ x2 + ( y2 - 1 ) * size ], db / dheight * w );
                atomicAdd( &tmpWater[ x2 + 1 + ( y2 - 1 ) * size ], dbr / dheight * w );

                water[ indexGlobal ] = 1.0f + remainingWater;
            }

            dheightV[ indexGlobal ] = dheight;
    //    }
    //}
}

__global__
void erodeSecondPassKernel( float* __restrict__ heightmap, int size, float* __restrict__ water, float* __restrict__ tmpWater, float* __restrict__ dheightV )
{
    const float Scale = 1.0f;
    const float Erosion = 0.0005 * Scale;
    const float Deposition = 0.0000002 * Scale;

    int startx = threadIdx.x + blockIdx.x * blockDim.x;
    int starty = threadIdx.y + blockIdx.y * blockDim.y;
    int stridex = blockDim.x * gridDim.x;
    int stridey = blockDim.y * gridDim.y;

    for( int y = 1 + starty; y < size - 1; y += stridey )
    {
        for( int x = 1 + startx; x < size - 1; x += stridex )
        {
            int index = x + y * size;

            water[ index ] += tmpWater[ index ];
            tmpWater[ index ] = 0;

            float oldHeight = heightmap[ index ];
            heightmap[ index ] += ( -( dheightV[ index ] - 0.005f / Scale ) * water[ index ] ) * Erosion + water[ index ] * Deposition;

            if( oldHeight < heightmap[ index ] )
                water[ index ] = max( water[ index ] - ( heightmap[ index ] - oldHeight ) * 1000.0f, 0.0f );
        }
    }
}

void gpuErosion( int size, int iterations, bool multiPass )
{
    int bufferSize = sizeof( float ) * size * size;

    float* heightmap;
    hipMallocManaged( &heightmap, bufferSize );

    float* water;
    hipMallocManaged( &water, bufferSize );

    float* tmpWater;
    hipMallocManaged( &tmpWater, bufferSize );

    float* dheightV;
    hipMallocManaged( &dheightV, bufferSize );

    auto noise = generateNoiseTexture( size );

    for( unsigned int i = 0; i < noise.size(); i++ )
        heightmap[ i ] = noise[ i ];

    auto start = std::chrono::high_resolution_clock::now();

    if( multiPass )
    {
        for( int i = 0; i < iterations; i++ )
        {
            erodeFirstPassKernel<<< dim3( 64, 64 ), dim3( 32, 32 ) >>>( heightmap, size, water, tmpWater, dheightV );
            erodeSecondPassKernel<<< 4, dim3( 32, 32 ) >>>( heightmap, size, water, tmpWater, dheightV );
        }
    }
    else
    {
        erodeKernel<<< 4, dim3( 32, 32 ) >>>( heightmap, size, iterations, water, tmpWater, dheightV );
    }

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast< std::chrono::milliseconds >( end - start ).count();

    std::cout << "gpu erosion took " << duration << "ms" << std::endl;
    save( "noiseGPU.raw", size, size, std::vector< float >( heightmap, heightmap + size * size ) );

    hipFree( heightmap );
    hipFree( water );
    hipFree( tmpWater );
    hipFree( dheightV );
}

int main()
{
    const int HeightmapSize = 2048;
    const int Iterations = 300;
    //cpuErosion( HeightmapSize, Iterations );
    gpuErosion( HeightmapSize, Iterations, true );
    return 0;
}